#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <time.h>
#include "g_kernel.h"
#include <complex>

__global__ 
void g_kernel(hipFloatComplex w, hipFloatComplex gamma, hipFloatComplex* quasi_k, hipFloatComplex* delta_k, hipFloatComplex* k_weight, hipFloatComplex* g_wk, const int N)
{
    int index=threadIdx.x+blockIdx.x*blockDim.x;
    if(index<N){
        hipFloatComplex wg = hipCaddf(w, gamma);
        hipFloatComplex t1 = hipCdivf(hipCmulf(delta_k[index], delta_k[index]), (hipCaddf(wg, quasi_k[index])) );
        g_wk[index] = hipCdivf(k_weight[index], hipCsubf(hipCsubf(wg,t1), quasi_k[index])  );
    }
}

//template <class T>
void test_nate(int N){
    printf("******** Calling test_nate *************");
}

// //double call_g_kernel(double real_w, complex<double> gamma, double* real_quasi_k, double* real_delta_k, double* real_k_weight, const int N)
// template void
// float call_g_kernel(float w, std::complex<float> complex_gamma, float* real_quasi_k, float* real_delta_k, float* real_k_weight, const int N);

// template void
// double call_g_kernel(double w, std::complex<double> complex_gamma, double* real_quasi_k, double* real_delta_k, double* real_k_weight, const int N);



double call_g_kernel(double real_w, std::complex<double> complex_gamma, double* real_quasi_k, double* real_delta_k, double* real_k_weight, const int N)
{
    hipFloatComplex w, gamma;
    hipFloatComplex *quasi_k, *delta_k, *k_weight, *g_wk;
    hipFloatComplex *dev_quasi_k, *dev_delta_k, *dev_k_weight, *dev_g_wk;

    w = make_hipFloatComplex(100.0,0);
    gamma = make_hipFloatComplex(10.0,0);
 

    quasi_k = (hipFloatComplex*)malloc( sizeof(hipFloatComplex)*N);
    delta_k = (hipFloatComplex*)malloc( sizeof(hipFloatComplex)*N);
    k_weight = (hipFloatComplex*)malloc( sizeof(hipFloatComplex)*N);
    g_wk = (hipFloatComplex*)malloc( sizeof(hipFloatComplex)*N);
    hipMalloc((void**)&dev_quasi_k, sizeof(hipFloatComplex)*N);
    hipMalloc((void**)&dev_delta_k, sizeof(hipFloatComplex)*N);
    hipMalloc((void**)&dev_k_weight, sizeof(hipFloatComplex)*N);
    hipMalloc((void**)&dev_g_wk, sizeof(hipFloatComplex)*N);

    init_complex_array(quasi_k, N);
    init_complex_array(delta_k, N);
    init_complex_array(k_weight, N);
    init_complex_array(g_wk, N);

    hipMemcpy(dev_quasi_k, quasi_k, sizeof(hipFloatComplex)*N, hipMemcpyHostToDevice);
    hipMemcpy(dev_delta_k, delta_k, sizeof(hipFloatComplex)*N, hipMemcpyHostToDevice);
    hipMemcpy(dev_k_weight, k_weight, sizeof(hipFloatComplex)*N, hipMemcpyHostToDevice);
    hipMemcpy(dev_g_wk, g_wk, sizeof(hipFloatComplex)*N, hipMemcpyHostToDevice);
 
    g_kernel<<<N/256+1, 256>>>(w, gamma, dev_quasi_k, dev_delta_k, dev_k_weight, dev_g_wk, N);
    //printf("error code: %s\n",hipGetErrorString(hipGetLastError()));

    hipMemcpy(g_wk, dev_g_wk, sizeof(hipFloatComplex)*N, hipMemcpyDeviceToHost);

    hipFree(dev_delta_k);
    hipFree(dev_quasi_k);
    hipFree(dev_k_weight);
    hipFree(dev_g_wk);

    printf(">>>>>>>>>> final data:\n");
    print_complex_array(g_wk, N, "out-vector");

    return 0;
};

// template void
// float call_g_kernel(float w, std::complex<float> complex_gamma, float* real_quasi_k, float* real_delta_k, float* real_k_weight, const int N);

// template void
// double call_g_kernel(double w, std::complex<float> complex_gamma, double* real_quasi_k, double* real_delta_k, double* real_k_weight, const int N);


void init_complex_array(hipFloatComplex *a, const int N) {
    int i;
    int val = rand() % 4 + 1;
    for(i=0; i<N; i++)
        a[i] = make_hipFloatComplex(val,val);
}

void print_complex_array(hipFloatComplex *a, const int N, char *d){
    int i;
    for(i=0; i<N; i++)
        printf("\n%s[%d]: %f",d, i, hipCrealf(a[i]), hipCimagf(a[i]) );
    printf("\n");  
}
